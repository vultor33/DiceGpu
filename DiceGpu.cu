#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <fstream>
#include <vector>
#include <iostream>
#include <stdio.h>
#include <cmath>

#include "InitilizeMCAtoms.h"
#include "Fitness.h"
#include "Metropolis.h"
#include "Move.h"

using namespace std;

void printAtomsVectorDouble(vector<double> & atoms, string testName);

double calculateFitness(vector<double> &x, vector< vector<double> > &allEnergies);

double calculateFitnessI(vector<double> &x, vector< vector<double> > &allEnergies, int i);

int main()
{
	cout << "comecou" << endl;

	Metropolis metrop_;

	metrop_.startMetropolis();

	double Eold, Enew;
	
	int mcSteps = 100;

	Eold = calculateFitness(metrop_.allAtoms, metrop_.allEnergies);

	for (int i = 0; i < mcSteps; i++)
	{
		do // nAtoms iterations - one MC step
		{
			metrop_.move();
			Enew = calculateFitnessI(metrop_.allAtoms, metrop_.allEnergies, metrop_.getMolMoved());
			Eold = metrop_.propagate(Eold, Enew);
		} while (metrop_.getFinished());
		metrop_.resetMetropolis();
	}

	cout << "termino" << endl;
	cin.get();

    return 0;
}


double calculateFitness(vector<double> &x, vector< vector<double> > &allEnergies)
{
	Fitness fit_;
	double total = 0.0e0;
	// x1 x2 x3 ... y1 y2 y3 ... z1 z2 z3
	int natm = x.size() / 3;
	// j > i - triangular inferiror
	for (int i = 0; i < (natm - 1); i++)
	{
		for (int j = (i + 1); j < natm; j++)
		{
			allEnergies[i][j] = fit_.lennardJonesIJ(x, i, j, natm);
			total += allEnergies[i][j];
		}
	}

	if (isnan(total))
	{
		cout << "nam no total" << endl;
		exit(1);
	}
		


	return total;
}

double calculateFitnessI(
	vector<double> &x, 
	vector< vector<double> > &allEnergies,
	int i)
{
	Fitness fit_;
	double total = 0.0e0;
	// x1 x2 x3 ... y1 y2 y3 ... z1 z2 z3
	int natm = x.size() / 3;
	// j > i - triangular inferiror
	for (int j = 0; j < natm; j++)
	{
		if (i > j)
			allEnergies[j][i] = fit_.lennardJonesIJ(x, i, j, natm);
		else if (i < j)
			allEnergies[i][j] = fit_.lennardJonesIJ(x, i, j, natm);
	}

	for (int i = 0; i < (natm - 1); i++)
	{
		for (int j = (i + 1); j < natm; j++)
		{
			total += allEnergies[i][j];
		}
	}

	if (isnan(total))
	{
		cout << "nam no total" << endl;
		exit(1);
	}

	return total;
}


void printAtomsVectorDouble(vector<double> & atoms, string testName)
{
	int natm = atoms.size() / 3;
	ofstream teste_(testName.c_str());
	teste_ << natm << endl << "t" << endl;
	for (int i = 0; i < natm; i++)
	{
		teste_ << "H "
			<< atoms[i] << "  "
			<< atoms[i + natm] << "  "
			<< atoms[i + 2 * natm] << endl;
	}
	teste_.close();
}
