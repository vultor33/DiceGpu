
#include "hip/hip_runtime.h"
#include ""

#include <fstream>
#include <vector>
#include <iostream>
#include <stdio.h>

#include "InitilizeMCAtoms.h"

using namespace std;

void printAtomsVectorDouble(vector<double> & atoms, string testName)
{
	int natm = atoms.size() / 3;
	ofstream teste_(testName.c_str());
	teste_ << natm << endl << "t" << endl;
	for (int i = 0; i < natm; i++)
	{
		teste_ << "H "
			<< atoms[i] << "  "
			<< atoms[i + natm] << "  "
			<< atoms[i + 2 * natm] << endl;
	}
	teste_.close();
}

int main()
{
	cout << "comecou" << endl;

	InitilizeMCAtoms init_;
	init_.init();
	printAtomsVectorDouble(init_.getAtoms(), "test.xyz");

    return 0;
}
